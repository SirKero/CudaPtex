#include "hip/hip_runtime.h"
#include <iostream>
#include <Ptexture.h>
#include <memory>
#include <vector>
#include <stdint.h>
#include "ptex.hpp"
#include <algorithm>
#include <type_traits>

#include "unique_device_ptr.hpp"

#include <hip/hip_runtime.h>
#include <>
#include <math.h>


using namespace mufflon;

typedef uint8_t DATA_TYPE;

//Test Kernel. It returns an array with all read texels
__global__ void cudaTest2(DATA_TYPE* res, int faceId, int dimX, int dimY, cudaPtexture tex) {
	unsigned int x = threadIdx.x;
	unsigned int y = blockIdx.x;

	//Had to use specific conversion, if not result is wrong or NaN
	float u = __uint2float_rz(threadIdx.x);
	float v = __uint2float_rz(blockIdx.x);
	u /= __int2float_rz(dimX);
	v /= __int2float_rz(dimY);

	//Sample texel
	DATA_TYPE tmpArr[3];
	PtexelFetch(tmpArr, faceId, u, v, tex);

	//copy result to the returned array
	for (int i = 0; i < tex.numChannels; i++) {
		res[tex.numChannels * (x + dimX * y) + i] = tmpArr[i];
	}
}



int main(){

	int face = 176;	//FaceID, if bigger than numFaces, invalid results are printed


	//std::string filepath = "models/teapot/teapot.ptx";		//<<DT: uint8
	std::string filepath = "models/bunny/bunny.ptx";			//<<DT: uint8
	//std::string filepath = "models/triangle/triangle.ptx";	//<<DT: float

	//Fill the cuda Texture object
	cudaPtex pTexture;
	pTexture.loadFile(filepath.c_str(), cudaPtex::TextureType::dt_none, true);

	//Ptex texture as comparison
	Ptex::PtexTexture* texture;
	Ptex::String ptex_error;
	Ptex::PtexFilter* filter;
	texture = Ptex::PtexTexture::open(filepath.c_str(), ptex_error, true);
	if (texture == nullptr) {
		std::cout << "Error: Could not read ptex texture \n";
	}

	switch (texture->dataType())
	{
	case Ptex::DataType::dt_uint8:
		std::cout << "Ptex: Data Type is uint8 \n";
		break;
	case Ptex::DataType::dt_uint16:
		std::cout << "Ptex: Data Type is uint16 \n";
		break;
	case Ptex::DataType::dt_half:
		std::cout << "Ptex: Data Type is half \n";
		break;
	case Ptex::DataType::dt_float:
		std::cout << "Ptex: Data Type is float \n";
		break;
	}

	//Filter for correct u/v readings on triangles
	Ptex::PtexFilter::FilterType filterType = Ptex::PtexFilter::FilterType::f_point;
	Ptex::PtexFilter::Options opts(filterType);
	filter = Ptex::PtexFilter::getFilter(texture, opts);

	
	//Needed vars
	int texSize = texture->numChannels() * texture->getFaceInfo(face).res.size();
	int ResU = texture->getFaceInfo(face).res.u();
	int ResV = texture->getFaceInfo(face).res.v();
	float ResUF = static_cast<float>(ResU);
	float ResVF = static_cast<float>(ResV);
	int numChannels = pTexture.getNumChannels();
	int testFaceSize = numChannels * ResU * ResV;
	unique_device_ptr<Device::CUDA, DATA_TYPE[]> testRes = make_udevptr_array <Device::CUDA, DATA_TYPE, false>(testFaceSize);
	unique_device_ptr<Device::CPU, DATA_TYPE[]> cpuRes = make_udevptr_array <Device::CPU, DATA_TYPE, false>(testFaceSize);
	unique_device_ptr<Device::CPU, DATA_TYPE[]> cpuRes2 = make_udevptr_array <Device::CPU, DATA_TYPE, false>(testFaceSize);

	cudaTest2 << <ResV, ResU >> > (testRes.get(), face, ResU, ResV, pTexture.getTexture());

	hipMemcpy(cpuRes.get(), testRes.get(), testFaceSize * sizeof(DATA_TYPE), hipMemcpyDefault);

	//Prints the CUDA Samples. (0,0) is in the left top corner
	std::cout << "Cuda Sampled: \n\n";

	for (int y = 0; y < ResV; y++) {
		for (int x = 0; x < ResU; x++) {
			for (int i = 0; i < numChannels; i++) {
				int idx = numChannels * (x + y * ResU) + i;
				std::cout << +cpuRes[idx] << ",";
			}
		}
		std::cout << '\n';
	}

	//Prints the Ptex Samples. (0,0) is in the left top corner
	std::cout << "\n\n Ptex Sampled \n\n";

	for (int y = 0; y < ResV; y++) {
		for (int x = 0; x < ResU; x++) {
			float result[3];
			filter->eval(result,0,numChannels,face,static_cast<float>(x)/ResUF, static_cast<float>(y)/ResVF, 0.1f,0.0f,0.0f,0.1f);
			for (int i = 0; i < numChannels; i++) {
				DATA_TYPE resultU8;
				if (std::is_same<DATA_TYPE, float>::value) {
					resultU8 = static_cast<DATA_TYPE>(result[i]);
				}
				else if (std::is_same<DATA_TYPE, uint8_t>::value) {
					resultU8 = static_cast<DATA_TYPE>(result[i] * 255.0f);
				}
				std::cout << +resultU8 << ",";
				int idx = numChannels * (x + y * ResU) + i;
				cpuRes2[idx] = resultU8;
			}
			
		}
		std::cout << '\n';
	}

	//Prints the difference between both. Should be 0 or very small (float precision error)
	std::cout << "\n\n Max Diff \n\n";
	float diff = 0;

	for (int y = 0; y < ResV; y++) {
		for (int x = 0; x < ResU; x++) {
			diff = std::max(diff,static_cast<float>(std::abs(cpuRes[x + y * ResU] - cpuRes2[x + y * ResU])));
		}
	}

	std::cout << diff << '\n';


	texture->release();
}