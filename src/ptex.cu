#include "hip/hip_runtime.h"
/*
 PTEX SOFTWARE
 Copyright 2014 Disney Enterprises, Inc.  All rights reserved
 */

#include "ptex.hpp"
#include <iostream>
#include <math.h>
#include <memory>
#include <vector>
#include <type_traits>
#include <stdint.h>
//CUDA
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_fp16.h>


// Calculates the power for an int base and an uin8_t power.
__device__ 
int pow2(uint8_t pow) {
	int result = 1 << pow;
	return result;
}

//Cuda functions
__device__ 
void PtexelFetch(void* res, int faceIdx, float u, float v, int numChannels, void* texArr, 
	const uint32_t* texOffsetArr, const uint8_t* ResLog2U, const uint8_t* ResLog2V, cudaPtex::TextureType texType, bool isTriangle) {
	//calc Res U and Res V from the log2 variants from the array
	int ResU = pow2(ResLog2U[faceIdx]);
	int ResV = pow2(ResLog2V[faceIdx]);

	int offset = texOffsetArr[faceIdx];
	int index;
	if (!isTriangle) {
		 index = offset + ResU * numChannels * (u + v * ResV);
	}
	//For triangles: texture fetch after http://ptex.us/tritex.html
	else {
		float resf = __int2float_rz(ResU);		
		float ut = u * resf;
		float vt = v * resf;
		float uIdx = floorf(ut);
		float vIdx = floorf(vt);
		int tmpIndex;
		if ((ut - uIdx) + (vt - vIdx) <= 1.0f) {
			tmpIndex = __float2int_rz(uIdx + vIdx * resf);
		}
		else {
			tmpIndex = __float2int_rz((resf * resf - 1.0f) - (vIdx + uIdx * resf));
		}

		int iU = tmpIndex % ResU;
		int iV = tmpIndex / ResV;

		index = offset + numChannels * (iU + iV * ResU);

	}
	
	//Sample Data depending on type
	switch (texType)
	{
	case cudaPtex::dt_uint8:
		for (int i = 0; i < numChannels; i++) {
			reinterpret_cast<uint8_t*>(res)[i] = reinterpret_cast<uint8_t*>(texArr)[index + i];
		}
		break;
	case cudaPtex::dt_uint16:
		for (int i = 0; i < numChannels; i++) {
			reinterpret_cast<uint16_t*>(res)[i] = reinterpret_cast<uint16_t*>(texArr)[index + i];
		}
		break;
	case cudaPtex::dt_half:
		for (int i = 0; i < numChannels; i++) {
			reinterpret_cast<half*>(res)[i] = reinterpret_cast<half*>(texArr)[index + i];
		}
		break;
	case cudaPtex::dt_float:
		for (int i = 0; i < numChannels; i++) {
			reinterpret_cast<float*>(res)[i] = reinterpret_cast<float*>(texArr)[index + i];
		}
		break;
	}

	
}



__device__
void PtexelFetch(void* res, int faceIdx, float u, float v, cudaPtexture tex) {
	PtexelFetch(res, faceIdx, u, v,tex.numChannels, tex.data, tex.offset, tex.ResLog2U, tex.ResLog2V,tex.texType, tex.isTriangle);
}

void cudaPtex::loadFile(const char* filepath, bool premultiply) {
	
	//Load texture from file
	Ptex::PtexTexture* texture;
	Ptex::String ptexErr;
	texture = Ptex::PtexTexture::open(filepath, ptexErr, premultiply);
	//In experience this only triggers if the file is not a ptex file
	if (texture == nullptr) {
		std::cerr << "Ptex Error: " << ptexErr.c_str() << '\n';
		return;
	}

	//Get info about the texture
	m_numFaces = texture->numFaces();
	m_numChannels = texture->numChannels();
	
	if (texture->meshType() == Ptex::MeshType::mt_triangle) {
		m_isTriangle = true;
	}
	
	//Create CPU side Buffers
	uint32_t totalDataSize = 0;
	auto offsetBuf = std::make_unique<uint32_t[]>(m_numFaces);
	auto resUBuf = std::make_unique<uint8_t[]>(m_numFaces);
	auto resVBuf = std::make_unique<uint8_t[]>(m_numFaces);
	
	//Fill helpers from the Ptex FaceInfo
	offsetBuf[0] = 0;	//first one has no offset
	for (unsigned int i = 0; i < m_numFaces; i++) {
		Ptex::FaceInfo faceInfo = texture->getFaceInfo(i);
		resUBuf[i] = faceInfo.res.ulog2;
		resVBuf[i] = faceInfo.res.vlog2;
		totalDataSize += faceInfo.res.size() * m_numChannels;
		if (i < m_numFaces - 1) 
			offsetBuf[i + 1] = totalDataSize;	//offset for the data
		
	}

	uint32_t extraBufferSize = m_numFaces * sizeof(uint32_t) + 2 * m_numFaces * sizeof(uint8_t);

	
	uint32_t totalDataByteSize = 0;	//DataSize in Bytes (is depending on the Texture Type)

	//Check in which Data type the ptex file is in an read accordingly
	switch (texture->dataType()) {
	case Ptex::DataType::dt_uint8: 
		readPtexture<uint8_t>(texture, totalDataSize, extraBufferSize);
		m_DataType = TextureType::dt_uint8;
		totalDataByteSize += totalDataSize * sizeof(uint8_t);
		break;

	case Ptex::DataType::dt_uint16: 
		readPtexture<uint16_t>(texture, totalDataSize, extraBufferSize);
		m_DataType = TextureType::dt_uint16;
		totalDataByteSize += totalDataSize * sizeof(uint16_t);
		break;

	case Ptex::DataType::dt_float: 
		readPtexture<float>(texture, totalDataSize, extraBufferSize);
		m_DataType = TextureType::dt_float;
		totalDataByteSize += totalDataSize * sizeof(float);
		break;

	case Ptex::DataType::dt_half: 
		readPtexture<half>(texture, totalDataSize, extraBufferSize);
		m_DataType = TextureType::dt_half;
		totalDataByteSize += totalDataSize * sizeof(half);
		break;
	}

	//Copy Data to the GPU
	
	m_offsetPtr = reinterpret_cast<uint32_t*>(& m_dataArr[totalDataByteSize]);
	hipMemcpy(m_offsetPtr, offsetBuf.get(), m_numFaces * sizeof(uint32_t), hipMemcpyDefault);
	m_resLog2UPtr = reinterpret_cast<uint8_t*>(&m_dataArr[totalDataByteSize  + m_numFaces * sizeof(uint32_t)]);
	hipMemcpy(m_resLog2UPtr, resUBuf.get(), m_numFaces * sizeof(uint8_t), hipMemcpyDefault); 
	m_resLog2VPtr = reinterpret_cast<uint8_t*>(&m_dataArr[totalDataByteSize + m_numFaces * sizeof(uint32_t) + m_numFaces * sizeof(uint8_t)]);
	hipMemcpy(m_resLog2VPtr, resVBuf.get(), m_numFaces * sizeof(uint8_t), hipMemcpyDefault);

	//Copy data to gpu
	m_totalDataSize = totalDataSize;

	//release Ptex texture, it is not needed anymore
	texture->release(); 
}

cudaPtexture cudaPtex::getTexture() {
	return cudaPtexture{ getDataPointer(), getOffsetPointer(), getResLog2U(), getResLog2V(), getNumChannels(),m_DataType, m_isTriangle };
}

//TODO: Support half type
template <typename T>
void cudaPtex::readPtexture<T>(Ptex::PtexTexture (*texture),int totalDataSize, int extraBufferSize) {
	static_assert(std::is_same<T,uint8_t>::value || std::is_same<T, uint16_t>::value || std::is_same<T, float>::value || std::is_same<T, half>::value, "Type is not supportet by Ptex");
	
	auto dataBuf = std::make_unique<T[]>(totalDataSize);
	uint64_t offset = 0; //Data offset for desArr
	for (int i = 0; i < texture->numFaces(); i++) {
		std::vector<T> faceDataBuffer;
		int texSize = texture->numChannels() * texture->getFaceInfo(i).res.size();
		faceDataBuffer.resize(texSize);
		texture->getData(i, faceDataBuffer.data(), 0);

		//copyData
		for (int j = 0; j < texSize; j++)
			dataBuf[j + offset] = faceDataBuffer[j];
		
		//add to offset
		offset += texSize;
	}

	//copy to GPU
	m_dataArr = make_udevptr_array<Device::CUDA, char, false>(totalDataSize * sizeof(T) + extraBufferSize);
	hipMemcpy(m_dataArr.get(), dataBuf.get(), totalDataSize * sizeof(T), hipMemcpyDefault);

}




