#include "hip/hip_runtime.h"
/*
 PTEX SOFTWARE
 Copyright 2014 Disney Enterprises, Inc.  All rights reserved
 */

#include "ptex.hpp"
#include <iostream>
#include <math.h>
#include <memory>
#include <vector>
#include <type_traits>
#include <stdint.h>
//CUDA
#include <hip/hip_runtime.h>
#include <>


// Calculates the power for an int base and an uin8_t power.
__device__ 
int powI(int base, uint8_t pow) {
	int result = base;
	for (int i = 1; i < pow; i++) {
		result *= base;
	}
	return result;
}

//Cuda functions
__device__ 
void PtexelFetch(float* res, int faceIdx, float u, float v, int numChannels, const float* texArr, 
	const uint32_t* texOffsetArr, const uint8_t* ResLog2U, const uint8_t* ResLog2V, bool isTriangle) {
	//calc Res U and Res V from the log2 variants from the array
	int ResU = powI(2, ResLog2U[faceIdx]);
	int ResV = powI(2, ResLog2V[faceIdx]);

	int offset = texOffsetArr[faceIdx];
	int index;
	if (!isTriangle) {
		 index = offset + ResU * numChannels * (u + v * ResV);
	}
	//For triangles: texture fetch after http://ptex.us/tritex.html
	else {
		float resf = __int2float_rz(ResU);		
		float ut = u * resf;
		float vt = v * resf;
		float uIdx = floorf(ut);
		float vIdx = floorf(vt);
		int tmpIndex;
		if ((ut - uIdx) + (vt - vIdx) <= 1.0f) {
			tmpIndex = __float2int_rz(uIdx + vIdx * resf);
		}
		else {
			tmpIndex = __float2int_rz((resf * resf - 1.0f) - (vIdx + uIdx * resf));
		}

		int iU = tmpIndex % ResU;
		int iV = tmpIndex / ResV;

		index = offset + numChannels * (iU + iV * ResU);

	}

	for (int i = 0; i < numChannels; i++) {
		res[i] = texArr[index + i];
	}
}



__device__
void PtexelFetch(float* res, int faceIdx, float u, float v, cudaPtexture tex) {
	PtexelFetch(res, faceIdx, u, v,tex.numChannels, tex.data, tex.offset, tex.ResLog2U, tex.ResLog2V, tex.isTriangle);
}

void cudaPtex::loadFile(const char* filepath, bool premultiply) {
	
	//Load texture from file
	Ptex::PtexTexture* texture;
	Ptex::String ptexErr;
	texture = Ptex::PtexTexture::open(filepath, ptexErr, premultiply);
	//In experience this only triggers if the file is not a ptex file
	if (texture == nullptr) {
		std::cerr << "Ptex Error: " << ptexErr.c_str() << '\n';
		return;
	}

	//Get info about the texture
	m_numFaces = texture->numFaces();
	m_numChannels = texture->numChannels();
	
	if (texture->meshType() == Ptex::MeshType::mt_triangle) {
		m_isTriangle = true;
	}
	
	//Create CPU side Buffers
	uint64_t totalDataSize = 0;
	auto offsetBuf = std::make_unique<uint32_t[]>(m_numFaces);
	auto resUBuf = std::make_unique<uint8_t[]>(m_numFaces);
	auto resVBuf = std::make_unique<uint8_t[]>(m_numFaces);
	
	//Fill helpers from the Ptex FaceInfo
	offsetBuf[0] = 0;	//first one has no offset
	for (int i = 0; i < m_numFaces; i++) {
		Ptex::FaceInfo faceInfo = texture->getFaceInfo(i);
		resUBuf[i] = faceInfo.res.ulog2;
		resVBuf[i] = faceInfo.res.vlog2;
		totalDataSize += faceInfo.res.size() * m_numChannels;
		if (i < m_numFaces - 1) 
			offsetBuf[i + 1] = totalDataSize;	//offset for the data
		
	}

	//Copy data to GPU
	m_offsets = make_udevptr_array < Device::CUDA, uint32_t, false>(m_numFaces);
	m_ResLog2U = make_udevptr_array < Device::CUDA, uint8_t, false>(m_numFaces);
	m_ResLog2V = make_udevptr_array < Device::CUDA, uint8_t, false>(m_numFaces);
	
	hipMemcpy(m_offsets.get(), offsetBuf.get(), sizeof(uint32_t) * m_numFaces, hipMemcpyDefault);
	hipMemcpy(m_ResLog2U.get(), resUBuf.get(), sizeof(uint8_t) * m_numFaces, hipMemcpyDefault);
	hipMemcpy(m_ResLog2V.get(), resVBuf.get(), sizeof(uint8_t) * m_numFaces, hipMemcpyDefault);

	//Release tmpBuffers
	offsetBuf.release();
	resUBuf.release();
	resVBuf.release();
	
	//tmpBuffer for dataArray
	auto dataBuf = std::make_unique<float[]>(totalDataSize);

	Ptex::DataType dataType = texture->dataType();

	//Check in which Data type the ptex file is in an read accordingly
	switch (texture->dataType()) {
	case Ptex::DataType::dt_uint8 :
		readPtexture<uint8_t>(dataBuf.get(), texture);
		break;
	case Ptex::DataType::dt_uint16 :
		readPtexture<uint16_t>(dataBuf.get(), texture);
		break;
	case Ptex::DataType::dt_float :
		readPtexture<float>(dataBuf.get(), texture);
		break;
	//TODO: support half data type
	default:
		std::cerr << "Ptex Error: half Data Type is not supported";
	}

	//Copy data to gpu
	m_totalDataSize = totalDataSize;
	m_data = make_udevptr_array<Device::CUDA, float, false>(totalDataSize);

	hipMemcpy(m_data.get(), dataBuf.get(), totalDataSize * sizeof(float), hipMemcpyDefault);

	//release Ptex texture, it is not needed anymore
	texture->release(); 
}


//TODO: Support half type
template <typename T>
void cudaPtex::readPtexture<T>(float* desArr, Ptex::PtexTexture (*texture)) {
	static_assert(std::is_same<T,uint8_t>::value || std::is_same<T, uint16_t>::value || std::is_same<T, float>::value, "Ptex has a not supported type");

	uint64_t offset = 0; //Data offset for desArr
	for (int i = 0; i < texture->numFaces(); i++) {
		std::vector<T> faceDataBuffer;
		int texSize = texture->numChannels() * texture->getFaceInfo(i).res.size();
		faceDataBuffer.resize(texSize);
		texture->getData(i, faceDataBuffer.data(), 0);

		//if it is a float it can be copied
		if (std::is_same<T, float>::value) {	
			for (int j = 0; j < texSize; j++) 
				desArr[j + offset] = faceDataBuffer[j];
		}
		//uint8 and uint16 has to be converted to the range from 0 to 1
		else {
			T max = std::numeric_limits<T>::max();	//max val of uint8 and uint16 for division
			float maxf = static_cast<float>(max);
			for (int j = 0; j < texSize; j++) 
				desArr[j + offset] = static_cast<float>(faceDataBuffer[j]) / maxf;
		}
		//add to offset
		offset += texSize;
	}
}




